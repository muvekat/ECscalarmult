#include "hip/hip_runtime.h"
#include "prime_curve_arithmetic.cuh"
#include "curve_definitions.cuh"
#include "scalar_multiplication.cuh"

#define min(a,b) a<b?a:b

namespace ScalarMultiply {

	namespace pca = PrimeCurveArithmetic;

	/*
	Helper function used inside scalarMultiplyOnGpu.
	*/
	template <size_t N>
	__host__ bool launchScalarMultKernel(AffinePoint<N> *dev_res, BigNum<N> *dev_scalar,
		AffinePoint<N> *dev_point, CurveType curve, unsigned int count);


	//Using manual definitions so that curve can be inlined during compile time.
	__global__ void scalarMultKernelSecp192r1(AffinePoint<6> *res, BigNum<6> *scalar,
		AffinePoint<6> *point, unsigned int count)
	{
		for (unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count; idx += gridDim.x*blockDim.x) {
			res[idx] = pca::scalarMultBinary(scalar[idx], point[idx], &Curve::secp192r1);
		}
	}
	__global__ void scalarMultKernelSecp224r1(AffinePoint<7> *res, BigNum<7> *scalar,
		AffinePoint<7> *point, unsigned int count)
	{
		for (unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count; idx += gridDim.x*blockDim.x) {
			res[idx] = pca::scalarMultBinary(scalar[idx], point[idx], &Curve::secp224r1);
		}
	}
	__global__ void scalarMultKernelSecp256r1(AffinePoint<8> *res, BigNum<8> *scalar,
		AffinePoint<8> *point, unsigned int count)
	{
		for (unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count; idx += gridDim.x*blockDim.x) {
			res[idx] = pca::scalarMultBinary(scalar[idx], point[idx], &Curve::secp256r1);
		}
	}
	__global__ void scalarMultKernelSecp384r1(AffinePoint<12> *res, BigNum<12> *scalar,
		AffinePoint<12> *point, unsigned int count)
	{
		for (unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count; idx += gridDim.x*blockDim.x) {
			res[idx] = pca::scalarMultBinary(scalar[idx], point[idx], &Curve::secp384r1);
		}
	}
	__global__ void scalarMultKernelSecp521r1(AffinePoint<17> *res, BigNum<17> *scalar,
		AffinePoint<17> *point, unsigned int count)
	{
		for (unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < count; idx += gridDim.x*blockDim.x) {
			res[idx] = pca::scalarMultBinary(scalar[idx], point[idx], &Curve::secp521r1);
		}
	}


	//Manual definitions to bypass compile time errors of incompatible arguments.
	__host__ bool launchScalarMultKernel(AffinePoint<6> *dev_res, BigNum<6> *dev_scalar,
		AffinePoint<6> *dev_point, CurveType curve, unsigned int count)
	{
		const int threadCount = 256;
		const int blockCount = min(count/threadCount+1,256);

		switch (curve)
		{
		case secp192r1:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(scalarMultKernelSecp192r1), hipFuncCachePreferL1);
			scalarMultKernelSecp192r1 << <blockCount, threadCount >> > (dev_res, dev_scalar, dev_point, count);
			break;
		default:
			fprintf(stderr, "No curve type found!/n");
			return 1;
		}

		return 0;
	}
	__host__ bool launchScalarMultKernel(AffinePoint<7> *dev_res, BigNum<7> *dev_scalar,
		AffinePoint<7> *dev_point, CurveType curve, unsigned int count)
	{
		const int threadCount = 256;
		const int blockCount = min(count / threadCount + 1, 256);

		switch (curve)
		{
		case secp224r1:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(scalarMultKernelSecp224r1), hipFuncCachePreferL1);
			scalarMultKernelSecp224r1 << <blockCount, threadCount >> > (dev_res, dev_scalar, dev_point, count);
			break;
		default:
			fprintf(stderr, "No curve type found!/n");
			return 1;
		}

		return 0;
	}
	__host__ bool launchScalarMultKernel(AffinePoint<8> *dev_res, BigNum<8> *dev_scalar,
		AffinePoint<8> *dev_point, CurveType curve, unsigned int count)
	{
		const int threadCount = 256;
		const int blockCount = min(count / threadCount + 1, 256);

		switch (curve)
		{
		case secp256r1:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(scalarMultKernelSecp256r1), hipFuncCachePreferL1);
			scalarMultKernelSecp256r1 << <blockCount, threadCount >> > (dev_res, dev_scalar, dev_point, count);
			break;
		default:
			fprintf(stderr, "No curve type found!/n");
			return 1;
		}

		return 0;
	}
	__host__ bool launchScalarMultKernel(AffinePoint<12> *dev_res, BigNum<12> *dev_scalar,
		AffinePoint<12> *dev_point, CurveType curve, unsigned int count)
	{
		const int threadCount = 256;
		const int blockCount = min(count / threadCount + 1, 256);

		switch (curve)
		{
		case secp384r1:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(scalarMultKernelSecp384r1), hipFuncCachePreferL1);
			scalarMultKernelSecp384r1 << <blockCount, threadCount >> > (dev_res, dev_scalar, dev_point, count);
			break;
		default:
			fprintf(stderr, "No curve type found!/n");
			return 1;
		}

		return 0;
	}
	__host__ bool launchScalarMultKernel(AffinePoint<17> *dev_res, BigNum<17> *dev_scalar,
		AffinePoint<17> *dev_point, CurveType curve, unsigned int count)
	{
		const int threadCount = 256;
		const int blockCount = min(count / threadCount + 1, 256);

		switch (curve)
		{
		case secp521r1:
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(scalarMultKernelSecp521r1), hipFuncCachePreferL1);
			scalarMultKernelSecp521r1 << <blockCount, threadCount >> > (dev_res, dev_scalar, dev_point, count);
			break;
		default:
			fprintf(stderr, "No curve type found!/n");
			return 1;
		}

		return 0;
	}


	template <size_t N>
	__host__ hipError_t scalarMultiplyOnGpu(AffinePoint<N> *resultArray, BigNum<N> *scalarArray,
		AffinePoint<N> *pointArray, unsigned int count, CurveType curveType)
	{
		AffinePoint<N> *dev_res = 0;
		BigNum<N> *dev_scalar = 0;
		AffinePoint<N> *dev_point = 0;
		hipError_t cudaStatus = hipSuccess;

		//Allocate device memory.
		cudaStatus = hipMalloc((void**)&dev_point, count * sizeof(AffinePoint<N>));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_res, count * sizeof(AffinePoint<N>));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_scalar, count * sizeof(BigNum<N>));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		//Copy input arrays to device.
		cudaStatus = hipMemcpy(dev_point, pointArray, count * sizeof(AffinePoint<N>), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_scalar, scalarArray, count * sizeof(BigNum<N>), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		//Check if kernel launch returns non zero.
		if (launchScalarMultKernel(dev_res, dev_scalar, dev_point, curveType, count)) {
			goto Error;
		}

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "launchScalarMultKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		//Copy result to output array.
		cudaStatus = hipMemcpy(resultArray, dev_res, count * sizeof(AffinePoint<N>), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy devicetohost launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

	Error:
		//Free pointers.
		hipFree(dev_point);
		hipFree(dev_scalar);
		hipFree(dev_res);

		return cudaStatus;
	}
}

//Exposing for linkage only supported scalarMultiplyOnGpu.
template hipError_t ScalarMultiply::scalarMultiplyOnGpu(AffinePoint<6> *resultArray, BigNum<6> *scalarArray,
	AffinePoint<6> *pointArray, unsigned int count, CurveType curveType);
template hipError_t ScalarMultiply::scalarMultiplyOnGpu(AffinePoint<7> *resultArray, BigNum<7> *scalarArray,
	AffinePoint<7> *pointArray, unsigned int count, CurveType curveType);
template hipError_t ScalarMultiply::scalarMultiplyOnGpu(AffinePoint<8> *resultArray, BigNum<8> *scalarArray,
	AffinePoint<8> *pointArray, unsigned int count, CurveType curveType);
template hipError_t ScalarMultiply::scalarMultiplyOnGpu(AffinePoint<12> *resultArray, BigNum<12> *scalarArray,
	AffinePoint<12> *pointArray, unsigned int count, CurveType curveType);
template hipError_t ScalarMultiply::scalarMultiplyOnGpu(AffinePoint<17> *resultArray, BigNum<17> *scalarArray,
	AffinePoint<17> *pointArray, unsigned int count, CurveType curveType);